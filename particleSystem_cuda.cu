#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// This file contains C wrappers around the some of the CUDA API and the
// kernel functions so that they can be called from "particleSystem.cpp"

#include <cstdlib>
#include <cstdio>
#include <string.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#include <helper_functions.h>
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "particles_kernel_impl.cuh"

extern "C"
{

    void cudaInit(int argc, char **argv)
    {
        int devID;

        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        devID = findCudaDevice(argc, (const char **)argv);

        if (devID < 0)
        {
            printf("No CUDA Capable devices found, exiting...\n");
            exit(EXIT_SUCCESS);
        }
    }

    void allocateArray(void **devPtr, size_t size)
    {
        checkCudaErrors(hipMalloc(devPtr, size));
    }

    void freeArray(void *devPtr)
    {
        checkCudaErrors(hipFree(devPtr));
    }

    void threadSync()
    {
        checkCudaErrors(hipDeviceSynchronize());
    }

	void copyArrayToDevice(void *device, const void *host, int offset, int size)
    {
        checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
    }

	void copyArrayFromDevice(void *host, const void *device, int size)
    {
        checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
    }

    void setParameters(SimParams *hostParams)
    {
        // copy parameters to constant memory
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)));
    }

    //Round a / b to nearest higher integer value
    uint iDivUp(uint a, uint b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    // compute grid and thread block size for a given number of elements
    void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = 4000;
    }

    void integrateSystem(float *pos,
                         float *vel,
                         float *force,
                         float deltaTime,
                         uint numParticles)
    {
        thrust::device_ptr<float4> d_pos4((float4 *)pos);
        thrust::device_ptr<float4> d_vel4((float4 *)vel);
		thrust::device_ptr<float4> d_force4((float4 *)force);

        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4, d_force4)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_vel4+numParticles, d_force4+numParticles)),
            integrate_functor(deltaTime));
    }
    
    void vintegrateSystem(float *vel,
                         float *force,
                         float deltaTime,
                         uint numParticles)
    {
        thrust::device_ptr<float4> d_vel4((float4 *)vel);
		thrust::device_ptr<float4> d_force4((float4 *)force);

        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_vel4, d_force4)),
            thrust::make_zip_iterator(thrust::make_tuple(d_vel4+numParticles, d_force4+numParticles)),
            vintegrate_functor(deltaTime));
    }

    void calcHash(uint  *gridParticleHash,
                  uint  *gridParticleIndex,
                  float *pos,
                  int    numParticles)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // execute the kernel
        calcHashD<<< numBlocks, numThreads >>>(gridParticleHash,
                                               gridParticleIndex,
                                               (float4 *) pos,
                                               numParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }

    void reorderDataAndFindCellStart(uint  *cellStart,
                                     uint  *cellEnd,
                                     float *sortedPos,
                                     float *sortedVel,
                                     float *sortedForce,
                                     uint  *gridParticleHash,
                                     uint  *gridParticleIndex,
                                     float *oldPos,
                                     float *oldVel,
                                     float *oldForce,
                                     uint   numParticles,
                                     uint   numCells)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // set all cells to empty
        checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

        uint smemSize = sizeof(uint)*(numThreads+1);
        reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
            cellStart,
            cellEnd,
            (float4 *) sortedPos,
            (float4 *) sortedVel,
            (float4 *) sortedForce,
            gridParticleHash,
            gridParticleIndex,
            (float4 *) oldPos,
            (float4 *) oldVel,
            (float4 *) oldForce,
            numParticles);
        getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");

    }

    void collide(float *newForce,
                 float *sortedPos,
                 float *sortedVel,
                 float *sortedForce,
                 uint  *gridParticleIndex,
                 uint  *cellStart,
                 uint  *cellEnd,
                 uint   numParticles,
                 uint   numCells)
    {

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 64, numBlocks, numThreads);

        // execute the kernel
        collideD<<< numBlocks, numThreads >>>((float4 *)newForce,
                                              (float4 *)sortedPos,
                                              (float4 *)sortedVel,
                                              (float4 *)sortedForce,
                                              gridParticleIndex,
                                              cellStart,
                                              cellEnd,
                                              numParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

    }


    void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
    {
        thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
                            thrust::device_ptr<uint>(dGridParticleHash + numParticles),
                            thrust::device_ptr<uint>(dGridParticleIndex));
    }

}   // extern "C"
